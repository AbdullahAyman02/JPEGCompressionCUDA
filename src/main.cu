#include "hip/hip_runtime.h"
%%writefile 1200488.cu
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cassert>
#include <opencv2/opencv.hpp>
#include <hip/hip_cooperative_groups.h>
#include <chrono>
#include <sys/stat.h>
#include <fstream>

using namespace std;
using namespace cooperative_groups;
namespace cg = cooperative_groups;

// GPU Constants
__constant__ float d_LuminanceQuantTable[8][8];
__constant__ float d_ChrominanceQuantTable[8][8];
__constant__ float d_dctMatrix[8][8];
__constant__ float d_idctMatrix[8][8];
__constant__ int d_zigzagOrder[64];

// Compression Structures
struct GPURLEBlock
{
    int y_size, cb_size, cr_size;
    int16_t *y_data;
    int16_t *cb_data;
    int16_t *cr_data;
};

struct GPUCompressedData
{
    int width, height;
    int num_blocks;
    GPURLEBlock *blocks;
};

// Timing utilities
class GPUTimer
{
    hipEvent_t start, stop;

public:
    GPUTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }
    ~GPUTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }
    void record(hipEvent_t event) { hipEventRecord(event); }
    float elapsed(hipEvent_t start, hipEvent_t stop)
    {
        float ms;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&ms, start, stop);
        return ms;
    }
};

// Host Quantization Tables and DCT Matrices (converted to float)
static const float h_LuminanceQuantTable[8][8] = {
    {16.0f, 11.0f, 10.0f, 16.0f, 24.0f, 40.0f, 51.0f, 61.0f},
    {12.0f, 12.0f, 14.0f, 19.0f, 26.0f, 58.0f, 60.0f, 55.0f},
    {14.0f, 13.0f, 16.0f, 24.0f, 40.0f, 57.0f, 69.0f, 56.0f},
    {14.0f, 17.0f, 22.0f, 29.0f, 51.0f, 87.0f, 80.0f, 62.0f},
    {18.0f, 22.0f, 37.0f, 56.0f, 68.0f, 109.0f, 103.0f, 77.0f},
    {24.0f, 35.0f, 55.0f, 64.0f, 81.0f, 104.0f, 113.0f, 92.0f},
    {49.0f, 64.0f, 78.0f, 87.0f, 103.0f, 121.0f, 120.0f, 101.0f},
    {72.0f, 92.0f, 95.0f, 98.0f, 112.0f, 100.0f, 103.0f, 99.0f}};

static const float h_ChrominanceQuantTable[8][8] = {
    {17.0f, 18.0f, 24.0f, 47.0f, 99.0f, 99.0f, 99.0f, 99.0f},
    {18.0f, 21.0f, 26.0f, 66.0f, 99.0f, 99.0f, 99.0f, 99.0f},
    {24.0f, 26.0f, 56.0f, 99.0f, 99.0f, 99.0f, 99.0f, 99.0f},
    {47.0f, 66.0f, 99.0f, 99.0f, 99.0f, 99.0f, 99.0f, 99.0f},
    {99.0f, 99.0f, 99.0f, 99.0f, 99.0f, 99.0f, 99.0f, 99.0f},
    {99.0f, 99.0f, 99.0f, 99.0f, 99.0f, 99.0f, 99.0f, 99.0f},
    {99.0f, 99.0f, 99.0f, 99.0f, 99.0f, 99.0f, 99.0f, 99.0f},
    {99.0f, 99.0f, 99.0f, 99.0f, 99.0f, 99.0f, 99.0f, 99.0f}};

static const float h_dctMatrix[8][8] = {
    {0.35355338f, 0.35355338f, 0.35355338f, 0.35355338f, 0.35355338f, 0.35355338f, 0.35355338f, 0.35355338f},
    {0.49039264f, 0.41573481f, 0.27778512f, 0.09754516f, -0.09754516f, -0.27778512f, -0.41573481f, -0.49039264f},
    {0.46193975f, 0.19134172f, -0.19134172f, -0.46193975f, -0.46193975f, -0.19134172f, 0.19134172f, 0.46193975f},
    {0.41573481f, -0.09754516f, -0.49039264f, -0.27778512f, 0.27778512f, 0.49039264f, 0.09754516f, -0.41573481f},
    {0.35355338f, -0.35355338f, -0.35355338f, 0.35355338f, 0.35355338f, -0.35355338f, -0.35355338f, 0.35355338f},
    {0.27778512f, -0.49039264f, 0.09754516f, 0.41573481f, -0.41573481f, -0.09754516f, 0.49039264f, -0.27778512f},
    {0.19134172f, -0.46193975f, 0.46193975f, -0.19134172f, -0.19134172f, 0.46193975f, -0.46193975f, 0.19134172f},
    {0.09754516f, -0.27778512f, 0.41573481f, -0.49039264f, 0.49039264f, -0.41573481f, 0.27778512f, -0.09754516f}};

static const float h_idctMatrix[8][8] = {
    {0.35355338f, 0.49039264f, 0.46193975f, 0.41573481f, 0.35355338f, 0.27778512f, 0.19134172f, 0.09754516f},
    {0.35355338f, 0.41573481f, 0.19134172f, -0.09754516f, -0.35355338f, -0.49039264f, -0.46193975f, -0.27778512f},
    {0.35355338f, 0.27778512f, -0.19134172f, -0.49039264f, -0.35355338f, 0.09754516f, 0.46193975f, 0.41573481f},
    {0.35355338f, 0.09754516f, -0.46193975f, -0.27778512f, 0.35355338f, 0.41573481f, -0.19134172f, -0.49039264f},
    {0.35355338f, -0.09754516f, -0.46193975f, 0.27778512f, 0.35355338f, -0.41573481f, -0.19134172f, 0.49039264f},
    {0.35355338f, -0.27778512f, -0.19134172f, 0.49039264f, -0.35355338f, -0.09754516f, 0.46193975f, -0.41573481f},
    {0.35355338f, -0.41573481f, 0.19134172f, 0.09754516f, -0.35355338f, 0.49039264f, -0.46193975f, 0.27778512f},
    {0.35355338f, -0.49039264f, 0.46193975f, -0.41573481f, 0.35355338f, -0.27778512f, 0.19134172f, -0.09754516f}};

static const int zigzagOrder[64] = {
    0, 1, 5, 6, 14, 15, 27, 28,
    2, 4, 7, 13, 16, 26, 29, 42,
    3, 8, 12, 17, 25, 30, 41, 43,
    9, 11, 18, 24, 31, 40, 44, 53,
    10, 19, 23, 32, 39, 45, 52, 54,
    20, 22, 33, 38, 46, 51, 55, 60,
    21, 34, 37, 47, 50, 56, 59, 61,
    35, 36, 48, 49, 57, 58, 62, 63};

// GPU Kernels
__device__ float determineScale(int quality) {
    quality = max(1, min(quality, 100)); // Clamp quality to [1, 100]
    
    if (quality < 50) {
        return 50.0f / quality; // More aggressive quantization
    } else {
        return 2.1f - (quality * 2.0f / 100.0f); // Less quantization
    }
}

__device__ void zigzagScan(const int16_t *channel, int16_t *output) {
    for (int i = 0; i < 64; i++) {
        int row = d_zigzagOrder[i] / 8;
        int col = d_zigzagOrder[i] % 8;
        output[i] = channel[row * 8 + col];
    }
}

__device__ void rleEncode(const int16_t *zigzag, int16_t *output, int &size) {
    int16_t current = zigzag[0];
    int count = 1;
    size = 0;

    for (int i = 1; i < 64; i++) {
        if (zigzag[i] == current) {
            count++;
        } else {
            output[size++] = count;
            output[size++] = current;
            current = zigzag[i];
            count = 1;
        }
    }
    output[size++] = count;
    output[size++] = current;
}

//------------------------------------------------------------------------------
// ——— COMPRESS KERNEL ————————————————————————————————————————————————
//------------------------------------------------------------------------------
__global__ void gpuCompressKernel(float *input, GPUCompressedData output, int quality)
{
    int block_idx = blockIdx.x;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    __shared__ float smem[8][8][3];
    __shared__ int16_t dct_coeffs[8][8][3];
    __shared__ int16_t zigzag[64][3];
    __shared__ int16_t rle_buffer[128];

    // Load block data into shared memory (parallel over threads)
    for (int c = 0; c < 3; c++) {
        int idx = block_idx * 192 + ty * 24 + tx * 3 + c;
        smem[ty][tx][c] = input[idx];
    }
    __syncthreads();

    float scale = determineScale(quality);

    // Parallel DCT and Quantization for each channel
    for (int channel = 0; channel < 3; channel++) {
        // Each thread computes one DCT coefficient
        float sum = 0.0f;
        for (int k = 0; k < 8; k++) {
            for (int l = 0; l < 8; l++) {
                sum += d_dctMatrix[ty][k] * smem[k][l][channel] * d_idctMatrix[l][tx];
            }
        }
        dct_coeffs[ty][tx][channel] = static_cast<int16_t>(roundf(sum));
        __syncthreads();

        // Quantization (parallel)
        float quant_step = (channel == 0 ? d_LuminanceQuantTable[ty][tx] : d_ChrominanceQuantTable[ty][tx]) * scale;
        if (quant_step < 1.0f) quant_step = 1.0f;
        dct_coeffs[ty][tx][channel] = static_cast<int16_t>(roundf(dct_coeffs[ty][tx][channel] / quant_step));
        __syncthreads();

        // Zigzag (parallel: 64 threads, i.e., 8x8 block)
        int tid = ty * 8 + tx;
        if (tid < 64) {
            int row = d_zigzagOrder[tid] / 8;
            int col = d_zigzagOrder[tid] % 8;
            zigzag[tid][channel] = dct_coeffs[row][col][channel];
        }
        __syncthreads();

        // RLE (sequential, one thread per channel)
        if (tx == 0 && ty == 0) {
            int size = 0;
            int16_t channel_zigzag[64];
            for (int i = 0; i < 64; ++i)
                channel_zigzag[i] = zigzag[i][channel];
            rleEncode(channel_zigzag, rle_buffer, size);
            if (channel == 0) {
                output.blocks[block_idx].y_size = size;
                for (int i = 0; i < size; i++)
                    output.blocks[block_idx].y_data[i] = rle_buffer[i];
            } else if (channel == 1) {
                output.blocks[block_idx].cb_size = size;
                for (int i = 0; i < size; i++)
                    output.blocks[block_idx].cb_data[i] = rle_buffer[i];
            } else {
                output.blocks[block_idx].cr_size = size;
                for (int i = 0; i < size; i++)
                    output.blocks[block_idx].cr_data[i] = rle_buffer[i];
            }
        }
        __syncthreads();
    }
}

//------------------------------------------------------------------------------
// ——— DECOMPRESS KERNEL —————————————————————————————————————————————
//------------------------------------------------------------------------------
__global__ void gpuDecompressKernel(float *output, GPUCompressedData input, int quality)
{
    int block_idx = blockIdx.x;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    __shared__ float smem[8][8][3];
    __shared__ int16_t zigzag[64];
    __shared__ float dct_coeffs[8][8];

    float scale = determineScale(quality);

    for (int channel = 0; channel < 3; channel++) {
        // RLE decode (sequential, one thread per channel)
        if (tx == 0 && ty == 0) {
            int idx = 0;
            int size = 0;
            int16_t* rle_data = nullptr;
            int rle_size = 0;
            if (channel == 0) {
                rle_data = input.blocks[block_idx].y_data;
                rle_size = input.blocks[block_idx].y_size;
            } else if (channel == 1) {
                rle_data = input.blocks[block_idx].cb_data;
                rle_size = input.blocks[block_idx].cb_size;
            } else {
                rle_data = input.blocks[block_idx].cr_data;
                rle_size = input.blocks[block_idx].cr_size;
            }
            for (int i = 0; i < rle_size; i += 2) {
                int count = rle_data[i];
                int16_t value = rle_data[i + 1];
                while (count-- && idx < 64)
                    zigzag[idx++] = value;
            }
        }
        __syncthreads();

        // Inverse zigzag (parallel)
        int tid = ty * 8 + tx;
        if (tid < 64) {
            int row = d_zigzagOrder[tid] / 8;
            int col = d_zigzagOrder[tid] % 8;
            dct_coeffs[row][col] = zigzag[tid];
        }
        __syncthreads();

        // Dequantization (parallel)
        if (tx < 8 && ty < 8) {
            float quant = (channel == 0 ? d_LuminanceQuantTable[ty][tx] : d_ChrominanceQuantTable[ty][tx]) * scale;
            dct_coeffs[ty][tx] *= quant;
        }
        __syncthreads();

        // IDCT (parallel)
        if (tx < 8 && ty < 8) {
            float sum = 0.0f;
            for (int k = 0; k < 8; k++)
                for (int l = 0; l < 8; l++)
                    sum += d_idctMatrix[ty][k] * dct_coeffs[k][l] * d_dctMatrix[l][tx];
            smem[ty][tx][channel] = sum;
        }
        __syncthreads();
    }

    // Write back to output (parallel)
    for (int c = 0; c < 3; c++) {
        int idx = block_idx * 192 + ty * 24 + tx * 3 + c;
        output[idx] = smem[ty][tx][c];
    }
}

// Host functions (initializeGPUConstants, save/load, etc. remain similar with corrections)
// [Rest of the code remains the same with corrections for channel handling and quantization tables]

void initializeGPUConstants() {
    hipMemcpyToSymbol(HIP_SYMBOL(d_LuminanceQuantTable), h_LuminanceQuantTable, sizeof(h_LuminanceQuantTable));
    hipMemcpyToSymbol(HIP_SYMBOL(d_ChrominanceQuantTable), h_ChrominanceQuantTable, sizeof(h_ChrominanceQuantTable));
    hipMemcpyToSymbol(HIP_SYMBOL(d_dctMatrix), h_dctMatrix, sizeof(h_dctMatrix));
    hipMemcpyToSymbol(HIP_SYMBOL(d_idctMatrix), h_idctMatrix, sizeof(h_idctMatrix));
    hipMemcpyToSymbol(HIP_SYMBOL(d_zigzagOrder), zigzagOrder, sizeof(zigzagOrder));
}

// [Remaining host functions are adjusted accordingly, ensuring correct channel assignments and data handling]

size_t getFileSize(const string &filename)
{
    struct stat stat_buf;
    int rc = stat(filename.c_str(), &stat_buf);
    return rc == 0 ? stat_buf.st_size : 0;
}

void saveCompressedData(const GPUCompressedData &data, const string &filename)
{
    ofstream file(filename, ios::binary);
    int width = data.width;
    int height = data.height;
    int num_blocks = data.num_blocks;

    file.write(reinterpret_cast<const char *>(&width), sizeof(int));
    file.write(reinterpret_cast<const char *>(&height), sizeof(int));
    file.write(reinterpret_cast<const char *>(&num_blocks), sizeof(int));

    GPURLEBlock *h_blocks = new GPURLEBlock[num_blocks];
    hipMemcpy(h_blocks, data.blocks, num_blocks * sizeof(GPURLEBlock), hipMemcpyDeviceToHost);

    for (int i = 0; i < num_blocks; i++)
    {
        GPURLEBlock &block = h_blocks[i];
        int16_t *h_y_data = new int16_t[block.y_size];
        int16_t *h_cb_data = new int16_t[block.cb_size];
        int16_t *h_cr_data = new int16_t[block.cr_size];

        hipMemcpy(h_y_data, block.y_data, block.y_size * sizeof(int16_t), hipMemcpyDeviceToHost);
        hipMemcpy(h_cb_data, block.cb_data, block.cb_size * sizeof(int16_t), hipMemcpyDeviceToHost);
        hipMemcpy(h_cr_data, block.cr_data, block.cr_size * sizeof(int16_t), hipMemcpyDeviceToHost);

        file.write(reinterpret_cast<const char *>(&block.y_size), sizeof(int));
        file.write(reinterpret_cast<const char *>(h_y_data), block.y_size * sizeof(int16_t));
        file.write(reinterpret_cast<const char *>(&block.cb_size), sizeof(int));
        file.write(reinterpret_cast<const char *>(h_cb_data), block.cb_size * sizeof(int16_t));
        file.write(reinterpret_cast<const char *>(&block.cr_size), sizeof(int));
        file.write(reinterpret_cast<const char *>(h_cr_data), block.cr_size * sizeof(int16_t));

        delete[] h_y_data;
        delete[] h_cb_data;
        delete[] h_cr_data;
    }
    delete[] h_blocks;
}

GPUCompressedData loadCompressedData(const string &filename)
{
    ifstream file(filename, ios::binary);
    GPUCompressedData data;

    file.read(reinterpret_cast<char *>(&data.width), sizeof(int));
    file.read(reinterpret_cast<char *>(&data.height), sizeof(int));
    file.read(reinterpret_cast<char *>(&data.num_blocks), sizeof(int));

    hipMalloc(&data.blocks, data.num_blocks * sizeof(GPURLEBlock));
    GPURLEBlock *h_blocks = new GPURLEBlock[data.num_blocks];

    for (int i = 0; i < data.num_blocks; i++)
    {
        GPURLEBlock block;
        file.read(reinterpret_cast<char *>(&block.y_size), sizeof(int));
        file.read(reinterpret_cast<char *>(&block.cb_size), sizeof(int));
        file.read(reinterpret_cast<char *>(&block.cr_size), sizeof(int));

        int16_t *h_y_data = new int16_t[block.y_size];
        int16_t *h_cb_data = new int16_t[block.cb_size];
        int16_t *h_cr_data = new int16_t[block.cr_size];

        file.read(reinterpret_cast<char *>(h_y_data), block.y_size * sizeof(int16_t));
        file.read(reinterpret_cast<char *>(h_cb_data), block.cb_size * sizeof(int16_t));
        file.read(reinterpret_cast<char *>(h_cr_data), block.cr_size * sizeof(int16_t));

        hipMalloc(&block.y_data, block.y_size * sizeof(int16_t));
        hipMalloc(&block.cb_data, block.cb_size * sizeof(int16_t));
        hipMalloc(&block.cr_data, block.cr_size * sizeof(int16_t));

        hipMemcpy(block.y_data, h_y_data, block.y_size * sizeof(int16_t), hipMemcpyHostToDevice);
        hipMemcpy(block.cb_data, h_cb_data, block.cb_size * sizeof(int16_t), hipMemcpyHostToDevice);
        hipMemcpy(block.cr_data, h_cr_data, block.cr_size * sizeof(int16_t), hipMemcpyHostToDevice);

        delete[] h_y_data;
        delete[] h_cb_data;
        delete[] h_cr_data;

        h_blocks[i] = block;
    }
    hipMemcpy(data.blocks, h_blocks, data.num_blocks * sizeof(GPURLEBlock), hipMemcpyHostToDevice);
    delete[] h_blocks;
    return data;
}

void splitIntoBlocks(const cv::Mat &image, vector<cv::Mat> &blocks)
{
    int blockSize = 8;
    int width = image.cols;
    int height = image.rows;

    for (int y = 0; y < height; y += blockSize)
    {
        for (int x = 0; x < width; x += blockSize)
        {
            cv::Rect roi(x, y, blockSize, blockSize);
            if (roi.x + roi.width > width)
                roi.width = width - roi.x;
            if (roi.y + roi.height > height)
                roi.height = height - roi.y;
            cv::Mat block = image(roi).clone();
            if (block.rows < blockSize || block.cols < blockSize)
            {
                cv::copyMakeBorder(block, block, 0, blockSize - block.rows, 0, blockSize - block.cols, cv::BORDER_CONSTANT, cv::Scalar(0));
            }
            blocks.push_back(block);
        }
    }
}

void assembleBlocks(cv::Mat &image, const vector<cv::Mat> &blocks, cv::Size imageSize)
{
    image.create(imageSize, CV_32FC3);
    int blockSize = 8;
    int currentBlock = 0;
    for (int y = 0; y < imageSize.height; y += blockSize)
    {
        for (int x = 0; x < imageSize.width; x += blockSize)
        {
            if (currentBlock >= blocks.size())
                break;
            cv::Rect roi(x, y, blockSize, blockSize);
            cv::Mat block = blocks[currentBlock++];
            if (roi.x + roi.width > imageSize.width)
                roi.width = imageSize.width - roi.x;
            if (roi.y + roi.height > imageSize.height)
                roi.height = imageSize.height - roi.y;
            block(cv::Rect(0, 0, roi.width, roi.height)).copyTo(image(roi));
        }
    }
}

int main(int argc, char **argv)
{
    auto total_start = std::chrono::high_resolution_clock::now();

    if (argc < 2 || argc > 3)
    {
        cerr << "Usage: " << argv[0] << " <image_path> [quality]" << endl;
        return -1;
    }
    const char *IMAGEPATH = argv[1];
    const int quality = argc == 3 ? stoi(argv[2]) : 75;

    initializeGPUConstants();
    GPUTimer timer;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    cv::Mat image = cv::imread(IMAGEPATH, cv::IMREAD_COLOR);
    if (image.empty())
    {
        cerr << "Error: Could not read the image." << endl;
        return -1;
    }
    cv::Mat ycbcr;
    cv::cvtColor(image, ycbcr, cv::COLOR_BGR2YCrCb);
    ycbcr.convertTo(ycbcr, CV_32FC3);

    vector<cv::Mat> blocks;
    splitIntoBlocks(ycbcr, blocks);
    int num_blocks = blocks.size();

    float *d_input, *d_output;
    hipMalloc(&d_input, num_blocks * 8 * 8 * 3 * sizeof(float));
    hipMalloc(&d_output, num_blocks * 8 * 8 * 3 * sizeof(float));

    for (int i = 0; i < num_blocks; i++)
    {
        hipMemcpy(d_input + i * 192, blocks[i].data, 192 * sizeof(float), hipMemcpyHostToDevice);
    }

    GPUCompressedData compressed;
    compressed.width = image.cols;
    compressed.height = image.rows;
    compressed.num_blocks = num_blocks;
    hipMalloc(&compressed.blocks, num_blocks * sizeof(GPURLEBlock));

    GPURLEBlock *h_blocks = new GPURLEBlock[num_blocks];
    for (int i = 0; i < num_blocks; i++)
    {
        hipMalloc(&h_blocks[i].y_data, 128 * sizeof(int16_t));
        hipMalloc(&h_blocks[i].cb_data, 128 * sizeof(int16_t));
        hipMalloc(&h_blocks[i].cr_data, 128 * sizeof(int16_t));
    }
    hipMemcpy(compressed.blocks, h_blocks, num_blocks * sizeof(GPURLEBlock), hipMemcpyHostToDevice);
    delete[] h_blocks;

    hipEventRecord(start);
    gpuCompressKernel<<<num_blocks, dim3(8, 8)>>>(d_input, compressed, quality);
    hipEventRecord(stop);
    float compress_time = timer.elapsed(start, stop);

    saveCompressedData(compressed, "compressed.gpu");

    hipEventRecord(start);
    gpuDecompressKernel<<<num_blocks, dim3(8, 8)>>>(d_output, compressed, quality);
    hipEventRecord(stop);
    float decompress_time = timer.elapsed(start, stop);

    vector<cv::Mat> reconstructed(num_blocks);
    for (int i = 0; i < num_blocks; i++)
    {
        reconstructed[i].create(8, 8, CV_32FC3);
        hipMemcpy(reconstructed[i].data, d_output + i * 192, 192 * sizeof(float), hipMemcpyDeviceToHost);
    }

    cv::Mat final_image;
    assembleBlocks(final_image, reconstructed, image.size());
    final_image.convertTo(final_image, CV_8UC3);
    cv::cvtColor(final_image, final_image, cv::COLOR_YCrCb2BGR);
    cv::imwrite("output.jpg", final_image);

    // Calculate stats PROPERLY
    size_t original_raw_size = image.total() * image.elemSize(); // W*H*3 bytes
    size_t compressed_size = getFileSize("compressed.gpu");

    cout << "=== Performance Results ===" << endl;
    cout << "Compression time: " << compress_time << " ms" << endl;
    cout << "Decompression time: " << decompress_time << " ms" << endl;
    cout << "Original RAW size: " << original_raw_size << " bytes" << endl;
    cout << "Compressed size: " << compressed_size << " bytes" << endl;
    cout << "Compression ratio: " << (float)original_raw_size / compressed_size << ":1" << endl;
    cout << "Space saving: "
         << (1.0 - (double)compressed_size / original_raw_size) * 100.0 << "%" << endl;

    hipFree(d_input);
    hipFree(d_output);
    // Additional cleanup for compressed data
    GPURLEBlock *h_cleanup = new GPURLEBlock[num_blocks];
    hipMemcpy(h_cleanup, compressed.blocks, num_blocks * sizeof(GPURLEBlock), hipMemcpyDeviceToHost);
    for (int i = 0; i < num_blocks; i++)
    {
        hipFree(h_cleanup[i].y_data);
        hipFree(h_cleanup[i].cb_data);
        hipFree(h_cleanup[i].cr_data);
    }
    delete[] h_cleanup;
    hipFree(compressed.blocks);

    auto total_end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> total_elapsed = total_end - total_start;
    cout << "Total elapsed time (including all steps): " << total_elapsed.count() << " ms" << endl;

    return 0;
}